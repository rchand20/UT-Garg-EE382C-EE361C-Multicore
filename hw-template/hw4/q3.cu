#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void odd_count(int *arr, unsigned int *d_count, int length)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= length)
  {
    return;
  }

  if (arr[idx] % 2 != 0)
  {
    atomicAdd(d_count, 1);
  }
}

__global__ void mark(int *arr, int *predicates, int length)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= length)
  {
    return;
  }
  predicates[idx] = arr[idx] % 2 ? 1 : 0;
}

__global__ void scan(int *output, int *predicates, int *sums, int n)
{
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int block = bid * n;

  extern __shared__ int buffer[];

  buffer[2 * tid] = predicates[block + (2 * tid)];
  buffer[2 * tid + 1] = predicates[block + (2 * tid) + 1];

  int offset = 1;
  for (int d = n >> 1; d > 0; d >>= 1)
  {
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      buffer[b] += buffer[a];
    }

    offset *= 2;
  }

  __syncthreads();

  if (tid == 0)
  {
    sums[bid] = buffer[n - 1];
    buffer[n - 1] = 0;
  }

  for (int d = 1; d < n; d *= 2)
  {
    offset >>= 1;
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      int t = buffer[a];
      buffer[a] = buffer[b];
      buffer[b] += t;
    }
  }

  __syncthreads();

  output[block + (2 * tid)] = buffer[2 * tid];
  output[block + (2 * tid) + 1] = buffer[2 * tid + 1];
}

__global__ void add(int *output, int length, int *n)
{
  int blockId = blockIdx.x;
  int tid = threadIdx.x;
  int block = blockId * length;

  output[block + tid] += n[blockId];
}

__global__ void compact(int *result, int *input, int *predicates, int *scanned, int length)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= length)
  {
    return;
  }

  if (predicates[idx] == 1)
  {
    int address = scanned[idx];
    result[address] = input[idx];
    if (result[address] == 5031)
    {
      printf("this index is fucking up: %d. related scanned[idx] is %d. the input at this index is %d\n", idx, scanned[idx], input[idx]);
    }
  }
}

int main()
{
  vector<int> data;
  ifstream infile;
  infile.open("inp.txt");

  if (infile.is_open())
  {
    while (infile.good())
    {
      char cNum[10];
      infile.getline(cNum, 256, ',');
      int num = atoi(cNum);
      data.push_back(num);
      // cout << num << " ";
    }
    infile.close();
  }
  else
  {
    cout << "Error opening file";
  }

  int size = data.size();
  int size1 = size * sizeof(int);

  int *d_output;
  int *d_input;
  int *d_predicates;
  int *d_result;
  int *d_dummy_blocks_sums;
  int *d_sums;
  int *d_inc;
  int *input_copy;
  unsigned int *d_count;

  int *output = (int *)malloc(size1);
  unsigned int *count = (unsigned int *)malloc(sizeof(unsigned int));

  int blocks = size / 1024;
  if (size % 1024 != 0)
  {
    blocks += 1;
  }
  const int sharedSize = 2 * 1024 * sizeof(int);

  hipMalloc((void **)&d_sums, blocks * sizeof(int));
  hipMalloc((void **)&d_count, sizeof(unsigned int));
  hipMalloc((void **)&d_inc, blocks * sizeof(int));
  hipMalloc((void **)&d_dummy_blocks_sums, blocks * sizeof(int));
  hipMalloc((void **)&d_output, size1);
  hipMalloc((void **)&d_input, size1);
  hipMalloc((void **)&input_copy, size1);
  hipMalloc((void **)&d_predicates, size1);
  hipMalloc((void **)&d_result, size1);

  hipMemcpy(d_input, data.data(), size1, hipMemcpyHostToDevice);
  hipMemcpy(d_count, 0, sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(input_copy, data.data(), size1, hipMemcpyHostToDevice);

  odd_count<<<blocks, 1024>>>(input_copy, d_count, size);
  mark<<<blocks, 1024>>>(d_input, d_predicates, size);
  scan<<<blocks, 512, sharedSize>>>(d_output, d_predicates, d_sums, 1024);
  scan<<<1, (blocks + 1) / 2, sharedSize>>>(d_inc, d_sums, d_dummy_blocks_sums, 1024);
  add<<<blocks, 1024>>>(d_output, 1024, d_inc);
  compact<<<size, 1024>>>(d_result, input_copy, d_predicates, d_output, size);

  hipMemcpy(count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy(output, d_result, *count * sizeof(int), hipMemcpyDeviceToHost);

  ofstream outfile;
  outfile.open("q3.txt");

  if (outfile.is_open())
  {

    for (int i = 0; i < *count; i++)
    {
      outfile << output[i] << ", ";
    }

    outfile.close();
  }
  else
  {
    cout << "Error opening file";
  }

  hipDeviceSynchronize();

  hipFree(d_output);
  hipFree(d_input);
  hipFree(d_sums);
  hipFree(d_count);
  hipFree(d_inc);
  hipFree(d_predicates);
  hipFree(d_dummy_blocks_sums);

  free(output);
  free(count);

  return 0;
}