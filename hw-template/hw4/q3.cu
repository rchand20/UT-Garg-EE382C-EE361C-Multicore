#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void mark(int *arr, int *predicates, int length)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= length)
    return;
  predicates[idx] = arr[idx] % 2 == 0 ? 0 : 1;
}

__global__ void block_scan(int *output, int *input, int *sums, int n)
{
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blockOff = bid * n;

  extern __shared__ int buffer[];

  buffer[2 * tid] = input[blockOff + (2 * tid)];
  buffer[2 * tid + 1] = input[blockOff + (2 * tid) + 1];

  int offset = 1;
  for (int d = n >> 1; d > 0; d >>= 1)
  {
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      buffer[b] += buffer[a];
    }

    offset *= 2;
  }

  __syncthreads();

  if (tid == 0)
  {
    sums[bid] = buffer[n - 1];
    buffer[n - 1] = 0;
  }

  for (int d = 1; d < n; d *= 2)
  {
    offset >>= 1;
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      int t = buffer[a];
      buffer[a] = buffer[b];
      buffer[b] += t;
    }
  }

  __syncthreads();

  output[blockOff + (2 * tid)] = buffer[2 * tid];
  output[blockOff + (2 * tid) + 1] = buffer[2 * tid + 1];
}

__global__ void sum_scan(int *output, int *input, int n, int power)
{
  extern __shared__ int temp[];

  int tid = threadIdx.x;

  if (tid < n)
  {
    temp[2 * tid] = input[2 * tid];
    temp[2 * tid + 1] = input[2 * tid + 1];
  }
  else
  {
    temp[2 * tid] = 0;
    temp[2 * tid + 1] = 0;
  }

  int offset = 1;
  for (int d = power >> 1; d > 0; d >>= 1)
  {
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      temp[b] += temp[a];
    }
    offset *= 2;
  }

  if (tid == 0)
  {
    temp[power - 1] = 0;
  }

  for (int d = 1; d < power; d *= 2)
  {
    offset >>= 1;
    __syncthreads();
    if (tid < d)
    {
      int a = offset * (2 * tid + 1) - 1;
      int b = offset * (2 * tid + 2) - 1;
      int t = temp[a];
      temp[a] = temp[b];
      temp[b] += t;
    }
  }

  __syncthreads();

  if (tid < n)
  {
    output[2 * tid] = temp[2 * tid];
    output[2 * tid + 1] = temp[2 * tid + 1];
  }
}

__global__ void add(int *output, int length, int *n)
{
  int blockId = blockIdx.x;
  int tid = threadIdx.x;
  int blockOffset = blockId * length;

  output[blockOffset + tid] += n[blockId];
}

__global__ void compact(int *result, int *input, int *predicates, int *output, int length)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= length)
  {
    return;
  }

  if (predicates[idx] == 1)
  {
    result[output[idx]] = input[idx];
  }
}

int nextPowerOfTwo(int x)
{
  int power = 1;
  while (power < x)
  {
    power *= 2;
  }
  return power;
}

int main()
{
  vector<int> data;
  ifstream infile;
  infile.open("inp.text");

  if (infile.is_open())
  {
    while (infile.good())
    {
      char cNum[10];
      infile.getline(cNum, 256, ',');
      int num = atoi(cNum);
      data.push_back(num);
      // cout << num << " ";
    }
    infile.close();
  }
  else
  {
    cout << "Error opening file";
  }

  int size = data.size();
  int size1 = size * sizeof(int);

  int *d_output;
  int *d_input;
  int *d_predicates;
  int *d_result;

  int *output = (int *)malloc(size1);

  hipMalloc((void **)&d_output, size1);
  hipMalloc((void **)&d_input, size1);
  hipMalloc((void **)&d_predicates, size1);
  hipMalloc((void **)&d_result, size1);

  hipMemcpy(d_input, data.data(), size1, hipMemcpyHostToDevice);
  mark<<<size / 1024 + 1, 1024>>>(d_input, d_predicates, size);
  hipDeviceSynchronize();

  int *d_sums;
  int *d_inc;

  int blocks = size / 1024;
  blocks += 1;
  int power = nextPowerOfTwo(blocks);
  const int sharedSize = 2 * 1024 * sizeof(int);
  hipMalloc((void **)&d_sums, blocks * sizeof(int));
  hipMalloc((void **)&d_inc, blocks * sizeof(int));

  block_scan<<<blocks, 512, sharedSize>>>(d_output, d_predicates, d_sums, 1024);
  sum_scan<<<1, blocks / 2, 2 * power * sizeof(int)>>>(d_inc, d_sums, blocks, power);
  add<<<blocks, 1024>>>(d_output, 1024, d_inc);
  compact<<<size / 1024 + 1, 1024>>>(d_result, d_input, d_predicates, d_output, size);
  hipMemcpy(output, d_result, size1, hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++)
  {
    cout << output[i] << ", ";
  }
  hipDeviceSynchronize();

  hipFree(d_output);
  hipFree(d_input);
  hipFree(d_sums);
  hipFree(d_inc);
  hipFree(d_predicates);

  free(output);
}