
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

#define N 1024	
#define BLOCK_SIZE 16

__global__ void arrMin2(int *min, int *A, int *size) {
	__shared__ int sharedMin;

	int tid = threadIdx.x;

	if(tid == 0) {
		sharedMin = 1000;
	}

	__syncthreads();

	int localMin = 1000;

	for(int i = blockIdx.x * blockDim.x + tid; i < *size; i += blockDim.x) {
		int val = A[i];

		if(localMin > val) {
			localMin = val;
		}
	}

	atomicMin(&sharedMin, localMin);
	__syncthreads();

	if(tid == 0) {
		min[blockIdx.x] = sharedMin;
	}
}

int main() {
    vector<int> data;
	int *A, *min;
	int *d_min, *d_A, *d_size;
	int size; 

    ifstream infile;
    infile.open("inp2.text");
	
	// Read file input and push to vector
    if (infile.is_open()) {
        while (infile.good()){
            char cNum[10];
            infile.getline(cNum, 256, ',');
            int num = atoi(cNum);
            data.push_back(num);
        }
		size = data.size() * sizeof(int);
        infile.close();
    }
    else {
        cout << "Error opening file";
    }
	
	// Alloc space for host copies 
	min = (int *)malloc(size);   
	A = (int *)malloc(size);

	// Alloc space for device copies
	hipMalloc((void **) &d_min, sizeof(int));
	hipMalloc((void **) &d_A, data.size() * sizeof(int));
	hipMalloc((void **) &d_size, sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_A, data.data(), size, hipMemcpyHostToDevice);
	int temp = data.size();
	hipMemcpy(d_size, &temp, sizeof(int), hipMemcpyHostToDevice);

	arrMin2<<<N/BLOCK_SIZE, BLOCK_SIZE >>>(d_min, d_A, d_size);	
	
	// Copy result back to host
	hipMemcpy(min, d_min, sizeof(int), hipMemcpyDeviceToHost);

	cout << "The min is " << min[0] << '\n';

	// Cleanup
	free(min); free(A);
	hipFree(d_min); hipFree(d_A);

	return 0;
}
