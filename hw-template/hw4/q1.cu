
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

#define N 1024
#define BLOCK_SIZE 16

__global__ void arrMin(int *min, int *A, int *size)
{
	__shared__ int sharedMin;

	int tid = threadIdx.x;

	if (tid == 0)
	{
		sharedMin = 1000;
	}

	__syncthreads();

	int localMin = 1000;

	for (int i = blockIdx.x * blockDim.x + tid; i < *size; i += blockDim.x)
	{
		int val = A[i];

		if (localMin > val)
		{
			localMin = val;
		}
	}

	atomicMin(&sharedMin, localMin);
	__syncthreads();

	if (tid == 0)
	{
		min[blockIdx.x] = sharedMin;
	}
}

__global__ void makeB(int *A, int *B, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
	{
		B[i] = A[i] % 10;
	}
}

int main()
{
	vector<int> data;
	int *A, *B, *min;
	int *d_min, *d_A, *d_B, *d_size;
	int size;

	ifstream infile;
	infile.open("inp.txt");

	// Read file input and push to vector
	if (infile.is_open())
	{
		while (infile.good())
		{
			char cNum[10];
			infile.getline(cNum, 256, ',');
			int num = atoi(cNum);
			data.push_back(num);
		}
		size = data.size() * sizeof(int);
		infile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	// Alloc space for host copies
	min = (int *)malloc(size);
	A = (int *)malloc(size);
	B = (int *)malloc(size);

	// Alloc space for device copies
	hipMalloc((void **)&d_min, sizeof(int));
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_size, sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_A, data.data(), size, hipMemcpyHostToDevice);
	int temp = data.size();
	hipMemcpy(d_size, &temp, sizeof(int), hipMemcpyHostToDevice);

	arrMin<<<data.size() / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_min, d_A, d_size);
	makeB<<<data.size() / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_A, d_B, temp);

	hipDeviceSynchronize();

	// Copy result back to host
	hipMemcpy(min, d_min, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	ofstream outfile;
	outfile.open("q1a.txt");

	if (outfile.is_open())
	{

		outfile << "The min is " << min[0] << '\n';

		outfile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	outfile.open("q1b.txt");
	if (outfile.is_open())
	{

		for (int i = 0; i < data.size(); ++i)
		{
			outfile << B[i] << ' ';
		}

		outfile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	cout << '\n';

	// Cleanup
	free(min);
	free(A);
	hipFree(d_min);
	hipFree(d_A);

	return 0;
}
