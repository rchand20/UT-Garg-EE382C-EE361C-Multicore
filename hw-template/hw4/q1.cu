
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

#define N 1024	
#define BLOCK_SIZE 16

__global__ void arrMin(int *min, int *A, int *size) {
	__shared__ int sharedMin;

	int tid = threadIdx.x;

	if(tid == 0) {
		sharedMin = 1000;
	}

	__syncthreads();

	int localMin = 1000;

	for(int i = blockIdx.x * blockDim.x + tid; i < *size; i += blockDim.x) {
		int val = A[i];

		if(localMin > val) {
			localMin = val;
		}
	}

	atomicMin(&sharedMin, localMin);
	__syncthreads();

	if(tid == 0) {
		min[blockIdx.x] = sharedMin;
	}
}

__global__ void makeB(int *A, int *B, int *size) {
	int tid = threadIdx.x;
	

	for(int i = blockIdx.x * blockDim.x + tid; i < *size; i += blockDim.x) {
		B[i] = A[i] % 10;
	}	

	__syncthreads();
}

int main() {
    vector<int> data;
	int *A, *B, *min;
	int *d_min, *d_A, *d_B, *d_size;
	int size; 

    ifstream infile;
    infile.open("inp.text");
	
	// Read file input and push to vector
    if (infile.is_open()) {
        while (infile.good()){
            char cNum[10];
            infile.getline(cNum, 256, ',');
            int num = atoi(cNum);
            data.push_back(num);
        }
		size = data.size() * sizeof(int);
        infile.close();
    }
    else {
        cout << "Error opening file";
    }
	
	// Alloc space for host copies 
	min = (int *)malloc(size);   
	A = (int *)malloc(size);
	B = (int *)malloc(size);

	// Alloc space for device copies
	hipMalloc((void **) &d_min, sizeof(int));
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_size, sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_A, data.data(), size, hipMemcpyHostToDevice);
	int temp = data.size();
	hipMemcpy(d_size, &temp, sizeof(int), hipMemcpyHostToDevice);

	arrMin<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_min, d_A, d_size);	
	makeB<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_size);
	
	// Copy result back to host
	hipMemcpy(min, d_min, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	cout << "The min is " << min[0] << '\n';

	for(int i = 0; i < data.size(); ++i) {
		cout << B[i] << ' ';
	}
	cout << '\n';

	// Cleanup
	free(min); free(A);
	hipFree(d_min); hipFree(d_A);

	return 0;
}
