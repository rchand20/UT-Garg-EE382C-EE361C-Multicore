
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 1024

__global__ void countEntriesPartA(int *A, int *B, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
	{
		int range = A[i] / 100;
		atomicAdd(&B[range], 1);
	}

	__syncthreads();
}

__global__ void countEntriesPartB(int *A, int *B2, int size)
{
	__shared__ int localB[10];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid == 0)
	{
		for (int j = 0; j < 10; j++)
			localB[j] = 0;
	}

	if (i < size)
	{
		int range = A[i] / 100;
		atomicAdd(&localB[range], 1);
	}

	__syncthreads();

	if (tid == 0)
	{
		for (int j = 0; j < 10; ++j)
		{
			atomicAdd(&B2[j], localB[j]);
			//B2[j] += localB[j];
		}
	}

	__syncthreads();
}

__global__ void countEntriesPartC(int *B, int *C, int size)
{
	extern __shared__ int smem[];
	int tid = threadIdx.x;

	if (tid < 10)
	{
		smem[2 * tid] = B[2 * tid];
		smem[2 * tid + 1] = B[2 * tid + 1];
	}

	else
	{
		smem[2 * tid] = 0;
		smem[2 * tid + 1] = 0;
	}

	int offset = 1;
	for (int d = size >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			smem[bi] += smem[ai];
		}
		offset *= 2;
	}

	if (tid == 0)
	{
		smem[size - 1] = 0;
	}

	for (int d = 1; d < size; d *= 2)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < d)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			int temp = smem[ai];
			smem[ai] = smem[bi];
			smem[bi] += temp;
		}
	}

	__syncthreads();

	C[2 * tid] = smem[2 * tid + 1];
	C[2 * tid + 1] = smem[2 * tid + 2];
	if ((2 * tid + 1) == 9)
	{
		C[2 * tid + 1] = smem[2 * tid + 1] + B[9];
	}
}

int main()
{
	vector<int> data;
	int *A, *B, *C;
	int *B2;
	int *d_A, *d_B, *d_B2, *d_C;
	int size;
	int Bsize = 10 * sizeof(int);

	ifstream infile;
	infile.open("inp.txt");

	// Read file input and push to vector
	if (infile.is_open())
	{
		while (infile.good())
		{
			char cNum[10];
			infile.getline(cNum, 256, ',');
			int num = atoi(cNum);
			data.push_back(num);
		}
		size = data.size() * sizeof(int);
		infile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	// Alloc space for host copies
	A = (int *)malloc(size);
	B = (int *)calloc(10, sizeof(int));
	B2 = (int *)calloc(10, sizeof(int));
	C = (int *)calloc(10, sizeof(int));

	// Alloc space for device copies
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, Bsize);
	hipMalloc((void **)&d_B2, Bsize);
	hipMalloc((void **)&d_C, Bsize);

	// Copy inputs to device
	hipMemcpy(d_A, data.data(), size, hipMemcpyHostToDevice);

	countEntriesPartA<<<data.size() / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_A, d_B, data.size());
	countEntriesPartB<<<data.size() / BLOCK_SIZE + 1, BLOCK_SIZE>>>(d_A, d_B2, data.size());
	countEntriesPartC<<<1, 5, 2 * 16 * sizeof(int)>>>(d_B, d_C, 16);
	// Copy result back to host
	hipMemcpy(B, d_B, Bsize, hipMemcpyDeviceToHost);
	hipMemcpy(B2, d_B2, Bsize, hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, Bsize, hipMemcpyDeviceToHost);

	// for (int i = 0; i < 10; ++i)
	// {
	// 	cout << B[i] << ' ';
	// }
	// cout << '\n';

	// for (int i = 0; i < 10; ++i)
	// {
	// 	cout << B2[i] << ' ';
	// }
	// cout << '\n';

	// for (int i = 0; i < 10; ++i)
	// {
	// 	cout << C[i] << ' ';
	// }
	// cout << '\n';

	ofstream outfile;
	outfile.open("q2a.txt");

	if (outfile.is_open())
	{

		for (int i = 0; i < 10; i++)
		{
			outfile << 	B[i] << ", ";
		}

		outfile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	outfile.open("q2b.txt");

	if (outfile.is_open())
	{

		for (int i = 0; i < 10; i++)
		{
			outfile << 	B2[i] << ", ";
		}

		outfile.close();
	}
	else
	{
		cout << "Error opening file";
	}


	outfile.open("q2c.txt");

	if (outfile.is_open())
	{

		for (int i = 0; i < 10; i++)
		{
			outfile << 	C[i] << ", ";
		}

		outfile.close();
	}
	else
	{
		cout << "Error opening file";
	}

	// Cleanup
	free(A);
	free(B);
	free(B2);
	free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_B2);
	hipFree(d_C);

	return 0;
}
